
#include <hip/hip_runtime.h>
/*
 * File: poisson3d_gpu.cpp
 * Project: AccFFT
 * Created by Amir Gholami on 12/23/2014
 * Contact: contact@accfft.org
 * Copyright (c) 2014-2015
 */
/*
 * Poisson solver CUDA kernel.
 * 
 * \author Pierre Kestener
 * \date June 15, 2015
 */

typedef double Complex[2];


// =======================================================
// =======================================================
/*
 * Poisson fourier filter (CUDA kernel).
 * Divide fourier coefficients by -(kx^2+ky^2+kz^2).
 */
__global__ 
void poisson_fourier_filter_kernel(Complex *data_hat, 
				   int N[3],      // global sizes
				   int isize[3],  // local  sizes
				   int istart[3],
				   int methodNb) 
{
  double NX = N[0];
  double NY = N[1];
  double NZ = N[2];
  
  double Lx = 1.0;
  double Ly = 1.0;
  double Lz = 1.0;

  double dx = Lx/NX;
  double dy = Ly/NY;
  double dz = Lz/NZ;

  // take care (direction reverse order for cuda)
  // cuda X dir maps k
  // cuda Y dir maps j
  unsigned int k = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;
  //unsigned int i = blockDim.z * blockIdx.z + threadIdx.z;

  if (k < isize[2] and j < isize[1]) {

    double ky = istart[1]+j;
    double kz = istart[2]+k;

    double kky = (double) ky;
    double kkz = (double) kz;

    if (ky>NY/2)
      kky -= NY;
    if (kz>NZ/2)
      kkz -= NZ;
    
    for (int i=0, index=j*isize[2]+k; 
	 i < isize[0]; 
	 i++, index += isize[1]*isize[2]) {
      
      double kx = istart[0]+i;  
      double kkx = (double) kx;
      
      if (kx>NX/2)
	kkx -= NX;
      
      double scaleFactor = 0.0;
      
      if (methodNb==0) {
	
	/*
	 * method 0 (from Numerical recipes)
	 */
	
	scaleFactor=2*( 
		       (cos(1.0*2*M_PI*kx/NX) - 1)/(dx*dx) + 
		       (cos(1.0*2*M_PI*ky/NY) - 1)/(dy*dy) + 
		       (cos(1.0*2*M_PI*kz/NZ) - 1)/(dz*dz) )*(NX*NY*NZ);
	
	
      } else if (methodNb==1) {
	
	/*
	 * method 1 (just from Continuous Fourier transform of 
	 * Poisson equation)
	 */
	scaleFactor=-4*M_PI*M_PI*(kkx*kkx + kky*kky + kkz*kkz)*NX*NY*NZ;
	
      }
      
      
      if (kx!=0 or ky!=0 or kz!=0) {
	data_hat[index][0] /= scaleFactor;
	data_hat[index][1] /= scaleFactor;
      } else { // enforce mean value is zero
	data_hat[index][0] = 0.0;
	data_hat[index][1] = 0.0;
      }
      
    } // end for i

  } // end if
  
} // poisson_fourier_filter_kernel

// =======================================================
// =======================================================
/*
 * Poisson fourier filter.
 * Divide fourier coefficients by -(kx^2+ky^2+kz^2).
 */
#define POISSON_FILTER_DIMX 16
#define POISSON_FILTER_DIMY 16
void poisson_fourier_filter_gpu(Complex *data_hat, 
				int N[3],
				int isize[3],
				int istart[3],
				int methodNb) 
{

  // take care of direction order reversed :
  // CUDA X dir maps isize[2]
  // CUDA Y dir maps isize[1]
  // isize[0] is sweeped inside kernel
  int blocksInX = (isize[2]+POISSON_FILTER_DIMX-1)/POISSON_FILTER_DIMX;
  int blocksInY = (isize[1]+POISSON_FILTER_DIMY-1)/POISSON_FILTER_DIMY;

  dim3 DimGrid(blocksInX, blocksInY, 1);
  dim3 DimBlock(POISSON_FILTER_DIMX, POISSON_FILTER_DIMY, 1);
  poisson_fourier_filter_kernel<<<DimGrid, DimBlock>>>(data_hat,
						       N,
						       isize,
						       istart,
						       methodNb);

} // poisson_fourier_filter_gpu
